#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void up_sweep_kernel(int rounded_length, int *array, int twod) {
    int twod1 = twod * 2;
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if (index + twod1 <= rounded_length) {
        array[index + twod1 - 1] += array[index + twod - 1];
    }
}

__global__ void down_sweep_kernel(int rounded_length, int *array, int twod) {
    int twod1 = twod * 2;
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if (index + twod1 <= rounded_length) {
        int t = array[index + twod - 1];
        array[index + twod - 1] = array[index + twod1 - 1];
        array[index + twod1 - 1] += t;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */
    const int threads_per_block = 512;
    int rounded_length = nextPow2(length);

    for (int twod = 1; twod < rounded_length; twod *= 2) {
        int twod1 = twod * 2;
        int blocks = ((rounded_length + twod1 - 1) / twod1 + threads_per_block - 1) / threads_per_block;
        up_sweep_kernel<<<blocks, threads_per_block>>>(rounded_length, device_data, twod);
    }

    int zero = 0;
    hipMemcpy(&device_data[rounded_length - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    for (int twod = rounded_length / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        int blocks = ((rounded_length + twod1 - 1) / twod1 + threads_per_block - 1) / threads_per_block;
        down_sweep_kernel<<<blocks, threads_per_block>>>(rounded_length, device_data, twod);
    }
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_peaks_kernel(int length, int *input, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index && index < length - 1 && input[index] > input[index - 1] && input[index] > input[index + 1]) {
        output[index] = 1;
    } else {
        output[index] = 0;
    }
}

__global__ void set_zeros_kernel(int length, int *array) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    array[index] = 0;
}

__global__ void write_ahead_kernel(int length, int *input, int *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int write_to_index = input[index] - 1;
    if (index > 0 && input[index] > input[index - 1]) {
        output[write_to_index] = index - 1;
    }
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */
    const int threads_per_block = 512;
    int blocks = (length + threads_per_block - 1) / threads_per_block;
    find_peaks_kernel<<<blocks, threads_per_block>>>(length, device_input, device_output);
    exclusive_scan(device_output, length);
    int count = 0;
    hipMemcpy(&count, device_output + length - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(device_input, device_output, length * sizeof(int), hipMemcpyHostToDevice);
    write_ahead_kernel<<<blocks, threads_per_block>>>(length, device_input, device_output);

    return count;
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
